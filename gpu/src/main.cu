#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <algorithm>
#include <thrust/complex.h>
#include "cuda_util.h"

//windows + visualstudio is a truckload of shit
#ifndef M_PI
  #define M_PI 3.14159265358979323846
#endif

typedef thrust::complex<float> comp;
#define ci comp(0,1)

//fft kernel
template<int DATASIZE>
__global__ void fftOvgu(comp* hdata) {
  //determine thread id
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  //shared memory
  __shared__ comp data[DATASIZE];

  //read data to shared menory by using reversed bitorder
  if (tid == 0) {
    data[0] = hdata[tid];
  } else if (tid == DATASIZE - 1) {
    data[DATASIZE - 1] = hdata[tid];
  } else {
    unsigned int DATA_SIZE = DATASIZE;
    unsigned int new_index = 0;
    int b;
    for (unsigned int i = 0; i < tid; i++) {
      b = DATA_SIZE / 2;
      while (b > 0) {
        if (new_index >= b) {
          new_index -= b;
        } else {
          new_index += b;
          break;
        }
        b /= 2;
      }
    }
    data[new_index] = hdata[tid];
  }

  //TODO: Sync threadblocks
  __syncthreads();

  //going up again and calculate ft
  unsigned int stride = 1;
  unsigned int block_size = 2;
  unsigned int m;
  comp a,b;
  comp quick_math;
  while (stride < DATASIZE) {
    m = tid % stride;
    printf("tid: %d = %d\n");
    quick_math = thrust::exp(comp(-2,0)*ci*comp(M_PI,0)*comp(m,0)/comp(block_size,0));
    if ((tid % block_size) < (block_size/2)) {
      printf("tid %d entered\n", tid);
      a = data[tid];
      b = data[tid+stride]*quick_math;
      data[tid] = a + b;
      data[tid+stride] = a - b;
    }    
    block_size*= 2;
    stride *= 2;
    __syncthreads();
  }

  //copy back shared memory
  hdata[tid] = data[tid];
}

//program entry point
int main(int /*argc*/, char** /*argv*/) {

  const int n = 8;
  //generate input data
  comp* data = (comp*) malloc(sizeof(comp)*n);
  for (int i = 0; i < n; i++) {
    data[i] = comp(i+1,0);
  }

  //check execution environnement
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    std::cout << "Hobo aint got no money for nvidia graphic card!" << std::endl;
    return EXIT_FAILURE;
  }

  //query the device properties
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  printDeviceProps(devProp);

  //set the device
  int device_handle = 0;
  hipSetDevice(device_handle);

  //init memory aand allocate device memory
  comp* data_device = nullptr;
  checkErrorsCuda( hipMalloc((void **) &data_device, sizeof(comp) * n));

  //copy device memory
  checkErrorsCuda( hipMemcpy( (void*) data_device, data, sizeof(comp) * n, hipMemcpyHostToDevice ));

  //determine thread layout
  const int MAX_THREADS_PER_BLOCK = devProp.maxThreadsPerBlock;
  int num_threads_per_block = std::min(n, MAX_THREADS_PER_BLOCK);
  int num_blocks = n/MAX_THREADS_PER_BLOCK;
  if( 0 != n % MAX_THREADS_PER_BLOCK) {
    num_blocks++;
  }
  std::cout << "num_blocks = " << num_blocks << "num_threads_per_block = " << num_threads_per_block << std::endl;


  //run kernel
  fftOvgu<n> <<< num_blocks, num_threads_per_block >>> (data_device); 
  
  //print result
  for (int i = 0; i < n; i++) {
    std::cout << data[i] << std::endl;
  }

  //copy result back
  checkErrorsCuda( hipMemcpy( data, data_device, sizeof(comp) * n, hipMemcpyDeviceToHost));

  //clean memory
  checkErrorsCuda( hipFree( data_device));
  free(data);
  return EXIT_SUCCESS;
}
