#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <algorithm>
#include "cuda_util.h"

//fft kernel
template<int DATASIZE>
__global__ void fftOvgu(float* hdata, const int hdata_size) {
  //determine thread id
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  //shared memory
  __shared__ float data[DATASIZE];

  //global counter to sync threadblocks
  __global__ int finished_threads = 0;

  //read data to shared menory by using reversed bitorder
  if (tid == 0) {
    data[0] = hdata[tid];
  } else if (tid == DATASIZE - 1) {
    data[DATASIZE - 1] = hdata[tid];
  } else {
    unsigned int DATA_SIZE = DATASIZE;
    unsigned int new_index = 0;
    int b;
    for (unsigned int i = 0; i < tid; i++) {
      b = DATA_SIZE / 2;
      while (b > 0) {
        if (new_index >= b) {
          new_index -= b;
        } else {
          new_index += b;
          break;
        }
        b /= 2;
      }
    }
    data[new_index] = hdata[tid];
  }

  //TODO: Sync threadblocks
  __syncthreads();

  //going up again and calculate ft

  //copy back shared memory
  hdata[tid] = data[tid];
}

//program entry point
int main(int /*argc*/, char** /*argv*/) {

  const int n = 8;
  //generate input data
  float* data = (float*) malloc(sizeof(float)*n);
  for (int i = 0; i < n; i++) {
    data[i] = (float) i+1;
  }

  //check execution environnement
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    std::cout << "Hobo aint got no money for nvidia graphic card!" << std::endl;
    return EXIT_FAILURE;
  }

  //query the device properties
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  printDeviceProps(devProp);

  //set the device
  int device_handle = 0;
  hipSetDevice(device_handle);

  //init memory aand allocate device memory
  float* data_device = nullptr;
  checkErrorsCuda( hipMalloc((void **) &data_device, sizeof(float) * n));

  //copy device memory
  checkErrorsCuda( hipMemcpy( (void*) data_device, data, sizeof(float) * n, hipMemcpyHostToDevice ));

  //determine thread layout
  const int MAX_THREADS_PER_BLOCK = devProp.maxThreadsPerBlock;
  int num_threads_per_block = std::min(n, MAX_THREADS_PER_BLOCK);
  int num_blocks = n/MAX_THREADS_PER_BLOCK;
  if( 0 != n % MAX_THREADS_PER_BLOCK) {
    num_blocks++;
  }
  std::cout << "num_blocks = " << num_blocks << "num_threads_per_block = " << num_threads_per_block << std::endl;


  //run kernel
  fftOvgu<n> <<< num_blocks, num_threads_per_block >>> (data_device, n); 
  
  //print result
  for (int i = 0; i < n; i++) {
    std::cout << data[i] << std::endl;
  }

  //copy result back
  checkErrorsCuda( hipMemcpy( data, data_device, sizeof(float) * n, hipMemcpyDeviceToHost));

  //clean memory
  checkErrorsCuda( hipFree( data_device));
  free(data);
  return EXIT_SUCCESS;
}
